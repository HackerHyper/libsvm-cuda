#include "hip/hip_runtime.h"
#include "CudaSolver.h"
#include "cuda_device_functions.h"
#include <memory>

CudaSolver *cudaSolver;

class CudaSolver::MinIdxFunctor // class object used for cross_block_reducer() template function
{
private:
	CValue_t *input_array, *output_array;
	int *input_idx, *output_idx;

public:
	MinIdxFunctor(CValue_t *obj_diff_array, int *obj_diff_idx, CValue_t *result_obj_min, int *result_idx) 
		: input_array(obj_diff_array), output_array(result_obj_min), input_idx(obj_diff_idx), output_idx(result_idx)
	{}

	void compute(size_t reduce_blocks, size_t reduce_block_size, int N) {
		size_t share_mem_size = reduce_block_size*(sizeof(CValue_t)+sizeof(int));
		cuda_find_min_idx<<<reduce_blocks, reduce_block_size, share_mem_size>>>(input_array, input_idx, output_array, output_idx, N);
	}

	void swap() {
		std::swap(input_array, output_array);
		std::swap(input_idx, output_idx);
	}

	int process_output() {
		// int Gmin_idx = -1; 
		// hipMemcpy(&Gmin_idx, &output_idx[0], sizeof(int), hipMemcpyDeviceToHost); // Gmin_idx should be in the first position now
		return -1;
	}
};

class CudaSolver::GmaxFunctor // class object used for cross_block_reducer() template function
{
private:
	GradValue_t *input_array1, *input_array2, *output_array1, *output_array2;
	int *input_idx, *output_idx;
	GradValue_t Gmax, Gmax2;
public:
	GmaxFunctor(GradValue_t *dh_gmax, GradValue_t *dh_gmax2, int *dh_gmax_idx, GradValue_t *result_gmax, GradValue_t *result_gmax2, int *result_gmax_idx)
		: input_array1(dh_gmax), input_array2(dh_gmax2), input_idx(dh_gmax_idx), output_array1(result_gmax), output_array2(result_gmax2), output_idx(result_gmax_idx)
	{}

	void compute(size_t reduce_blocks, size_t reduce_block_size, int N) {
		size_t share_mem_size = reduce_block_size*(2*sizeof(GradValue_t)+sizeof(int));
		cuda_find_gmax<<<reduce_blocks, reduce_block_size, share_mem_size>>>(input_array1, input_array2, input_idx, output_array1, output_array2, output_idx, N);
	}

	void swap() {
		std::swap(input_array1, output_array1);
		std::swap(input_array2, output_array2);
		std::swap(input_idx, output_idx);
	}

	int process_output() {
		int Gmax_idx = -1; 
		/* check_cuda_return("fail to copy output_idx from device",
		hipMemcpy(&Gmax_idx, &output_idx[0], sizeof(int), hipMemcpyDeviceToHost)); */ // Gmax_idx should be in the first position now
		CudaSolver::check_cuda_return("fail to copy output_array1 from device",
			hipMemcpy(&Gmax, &output_array1[0], sizeof(GradValue_t), hipMemcpyDeviceToHost));
		CudaSolver::check_cuda_return("fail to copy output_array2 from device",
			hipMemcpy(&Gmax2, &output_array2[0], sizeof(GradValue_t), hipMemcpyDeviceToHost));
		return Gmax_idx;
	}

	void get_gmax_values(GradValue_t &ret_Gmax, GradValue_t &ret_Gmax2)
	{
		ret_Gmax = Gmax;
		ret_Gmax2 = Gmax2;
	}
};

void CudaSolver::init_obj_diff_space(int l)
{
	dh_obj_diff_array = make_unique_cuda_array<CValue_t>(l);
	dh_obj_diff_idx = make_unique_cuda_array<int>(l);
	dh_result_obj_diff = make_unique_cuda_array<CValue_t>(num_blocks);
	dh_result_idx = make_unique_cuda_array<int>(num_blocks);
	return;
}

void CudaSolver::init_gmax_space(int l)
{
	dh_gmax = make_unique_cuda_array<GradValue_t>(l);
	dh_gmax2 = make_unique_cuda_array<GradValue_t>(l);
	dh_gmax_idx = make_unique_cuda_array<int>(l);
	dh_result_gmax = make_unique_cuda_array<GradValue_t>(num_blocks);
	dh_result_gmax2 = make_unique_cuda_array<GradValue_t>(num_blocks);
	dh_result_gmax_idx = make_unique_cuda_array<int>(num_blocks);
	return;
}

void CudaSolver::init_memory_arrays(int l) {
	int bsize = CUDA_BLOCK_SIZE; // TODO: query device for max thread block size
	while (l / bsize < 10 && bsize > 32) {
		bsize >>= 1; // halve it
	}

	block_size = bsize;
	num_blocks = l / block_size;
	if (l % block_size != 0) ++num_blocks;

	std::cout << "CUDA Integration\n";
	std::cout << "----------------\n";
	std::cout << "Selected thread block size:         " << bsize << std::endl;
	std::cout << "Selected number of blocks:          " << num_blocks << std::endl;
	std::cout << "Problem size:                       " << l << std::endl;
	std::cout << "Gradient vector stored as:          " << typeid(GradValue_t).name() << std::endl;

	result_idx.reset(new int[num_blocks]);
	result_obj_diff.reset(new CValue_t[num_blocks]);
	result_gmax.reset(new GradValue_t[num_blocks]);
	result_gmax2.reset(new GradValue_t[num_blocks]);

	init_obj_diff_space(l);
	init_gmax_space(l);
}

CudaSolver::CudaSolver(const svm_problem &prob, const svm_parameter &param)
	: eps(param.eps), kernel_type(param.kernel_type), svm_type(param.svm_type), mem_size(0)
{
	load_problem_parameters(prob, param);
}

CudaSolver::~CudaSolver()
{
	unbind_texture();
	hipDeviceReset();
}

void CudaSolver::compute_alpha()
{
	cuda_compute_alpha<<<1, 1>>>();
}

void CudaSolver::update_alpha_status()
{
	cuda_update_alpha_status<<<1,1>>>();
}

int CudaSolver::select_working_set_j(double Gmax, int &Gmin_idx, int l)
{

	cuda_compute_obj_diff<<<num_blocks, block_size>>>(Gmax, &dh_obj_diff_array[0], &dh_obj_diff_idx[0], l);

	MinIdxFunctor func(&dh_obj_diff_array[0], &dh_obj_diff_idx[0], &dh_result_obj_diff[0], &dh_result_idx[0]);
	Gmin_idx = cross_block_reducer(block_size, func, l);
	return Gmin_idx;
}


int CudaSolver::select_working_set(int &out_i, int &out_j, int l)
{
	GradValue_t Gmax = -GRADVALUE_MAX; // -INF;
	GradValue_t Gmax2 = -GRADVALUE_MAX; // -INF;
	int Gmax_idx = -1;
	int Gmin_idx = -1;

	cuda_prep_gmax<<<num_blocks, block_size, block_size*(2*sizeof(GradValue_t)+sizeof(int))>>>(&dh_gmax[0], &dh_gmax2[0], &dh_gmax_idx[0],  
		&dh_result_gmax[0], &dh_result_gmax2[0], &dh_result_gmax_idx[0], l);

	GmaxFunctor func(&dh_gmax[0], &dh_gmax2[0], &dh_gmax_idx[0], &dh_result_gmax[0], &dh_result_gmax2[0], &dh_result_gmax_idx[0]);
	Gmax_idx = cross_block_reducer(block_size, func, l);
	func.get_gmax_values(Gmax, Gmax2);

	dbgprintf(true, "Device: Gmax_idx %d Gmax %g Gmax2 %g\n", Gmax_idx, Gmax, Gmax2);

	if(Gmax+Gmax2 < eps)
		return 1;

	select_working_set_j(Gmax, Gmin_idx, l);

	out_i = Gmax_idx;
	out_j = Gmin_idx;

	return 0;
}

void CudaSolver::update_gradient(int l)
{
	cuda_update_gradient<<<num_blocks, block_size>>>(l);
}

void CudaSolver::fetch_vectors(double *G, double *alpha, char *alpha_status, int l)
{
	hipError_t err;
	{
		std::unique_ptr<GradValue_t[]> h_G(new GradValue_t[l]);
		err = hipMemcpy(&h_G[0], &dh_G[0], sizeof(GradValue_t) * l, hipMemcpyDeviceToHost);
		check_cuda_return("fail to copy from device dh_G", err);
		for (int i = 0; i < l; ++i)
			G[i] = h_G[i];
	}
	{
		std::unique_ptr<GradValue_t[]> h_alpha(new GradValue_t[l]);
		err = hipMemcpy(&h_alpha[0], &dh_alpha[0], sizeof(GradValue_t) * l, hipMemcpyDeviceToHost);
		check_cuda_return("fail to copy from device dh_alpha", err);
		for (int i = 0; i < l; ++i)
			alpha[i] = h_alpha[i];
	}

	err = hipMemcpy(alpha_status, &dh_alpha_status[0], sizeof(char) * l, hipMemcpyDeviceToHost);
	check_cuda_return("fail to copy from device dh_alpha_status", err);
}


void CudaSolver::setup_solver(const SChar_t *y, const double *QD, double *G, double *alpha, char *alpha_status, double Cp, double Cn, int l) 
{
	init_memory_arrays(l);
	
	hipError_t err;

	// allocate space for labels
	dh_y = make_unique_cuda_array<SChar_t>(l);

	err = hipMemcpy(&dh_y[0], y, sizeof(SChar_t) * l, hipMemcpyHostToDevice);
	check_cuda_return("fail to copy to device for dh_y", err);

	dh_QD = make_unique_cuda_array<CValue_t>(l);
	{
		std::unique_ptr<CValue_t[]> h_QD(new CValue_t[l]);
		for (int i = 0; i < l; ++i) {
			CHECK_FLT_RANGE(QD[i]);
			h_QD[i] = static_cast<CValue_t>(QD[i]);
		}

		err = hipMemcpy(&dh_QD[0], &h_QD[0], sizeof(CValue_t) * l, hipMemcpyHostToDevice);
		check_cuda_return("fail to copy to device for dh_QD", err);
	}

	/** allocate space for gradient vector */
	dh_G = make_unique_cuda_array<GradValue_t>(l);

	{
		std::unique_ptr<GradValue_t[]> h_G(new GradValue_t[l]);
		for (int i = 0; i < l; ++i)
			h_G[i] = static_cast<GradValue_t>(G[i]);

		err = hipMemcpy(&dh_G[0], &h_G[0], sizeof(GradValue_t) * l, hipMemcpyHostToDevice);
		check_cuda_return("fail to copy to device for dh_G", err);
	}

	dh_alpha = make_unique_cuda_array<GradValue_t>(l);

	{
		std::unique_ptr<GradValue_t[]> h_alpha(new GradValue_t[l]);
		for (int i = 0; i < l; ++i)
			h_alpha[i] = static_cast<GradValue_t>(alpha[i]);

		err = hipMemcpy(&dh_alpha[0], &h_alpha[0], sizeof(GradValue_t) * l, hipMemcpyHostToDevice);
		check_cuda_return("fail to copy to device for dh_alpha", err);
	}

	dh_alpha_status = make_unique_cuda_array<char>(l);

	hipMemcpy(&dh_alpha_status[0], alpha_status, sizeof(char) * l, hipMemcpyHostToDevice);
	check_cuda_return("fail to copy to device for dh_alpha_status", err);

	/** setup constants */
	err = update_solver_variables(&dh_y[0], &dh_QD[0],  &dh_G[0], &dh_alpha[0], &dh_alpha_status[0], Cp, Cn);

	check_cuda_return("fail to setup constants/textures", err);

	return ;
}

/**
Loads: kernel_type, svm_type, gamma, coef0, degree, x
*/
void CudaSolver::load_problem_parameters(const svm_problem &prob, const svm_parameter &param)
{
	hipError_t err;
	svm_node **x = prob.x;
	int l = prob.l;

	/** allocate space for support vectors */
	int elements = 0;
	for (int i = 0; i < l; ++i) 
	{
		const svm_node *tmp = x[i];
		while (tmp->index != -1) { // row terminator
			++elements; // count each row svm_node element
			++tmp;
		}
		++elements; // count the row terminating svm_node
	}

	/**
	NOTE: cuda_svm_node is typedef to float2
	float2.x == svm_node.index
	float2.y == svm_node.value
	*/
	std::unique_ptr<cuda_svm_node[]> x_space(new cuda_svm_node[elements]);
	for (int i = 0, j = 0; i < l; ++i) {
		const svm_node *tmp = x[i];
		while (tmp->index != -1) {
			x_space[j].x = static_cast<float>(tmp->index);
			x_space[j].y = static_cast<CValue_t>(tmp->value);
#ifdef DEBUG_VERIFY
			if (abs(tmp->value - x_space[j].y) > 1e-4) {
				std::cerr << "WARNING!: sample space value truncated by " 
					<< abs(tmp->value-x_space[j].y) << std::endl;
			}
#endif
			++j;
			++tmp;
		}
		x_space[j++].x = -1;
	}

	dh_space = make_unique_cuda_array<cuda_svm_node>(elements);

	err = hipMemcpy(&dh_space[0], &x_space[0], sizeof(cuda_svm_node) * elements, hipMemcpyHostToDevice);
	check_cuda_return("fail to copy to device for dh_space", err); 

	dh_x = make_unique_cuda_array<int>(l);

	{
		std::unique_ptr<int[]> h_x(new int[l]);

		int i = 0;
		bool assign_flag = false;
		for (int j = 0; j < elements; ++j)
		{
			if (!assign_flag) {
				if (i >= l) {
					throw std::runtime_error("error in updating h_x");
				}
				h_x[i] = j;
				assign_flag = true;
			}
			if (x_space[j].x == -1) {
				++i;
				assign_flag = false;
			}
		}

		err = hipMemcpy(&dh_x[0], &h_x[0], sizeof(int) * l, hipMemcpyHostToDevice);
		check_cuda_return("fail to copy to device for dh_x", err);
	}

	err = update_param_constants (param, &dh_x[0], &dh_space[0], sizeof(cuda_svm_node)*elements);
	check_cuda_return("fail to setup parameter constants", err);
}

void CudaSolver::setup_rbf_variables(double *x_square, int l)
{
	if (kernel_type != RBF)
		return ;

	/* x_square is only needed in computing the RBF kernel */
	std::unique_ptr<CValue_t[]> h_x_square(new CValue_t[l]);
	for (int i = 0; i < l; ++i)
		h_x_square[i] = static_cast<CValue_t>(x_square[i]);

	dh_x_square = make_unique_cuda_array<CValue_t>(l);

	hipError_t err = hipMemcpy(&dh_x_square[0], &h_x_square[0], sizeof(CValue_t) * l, hipMemcpyHostToDevice);
	check_cuda_return("fail to copy to device for dh_x_square", err);

	err = update_rbf_variables(&dh_x_square[0]);
	check_cuda_return("fail to update rbf variables", err);
}

void CudaSolver::show_memory_usage(const int &total_space)
{
	printf("Total space allocated on device:	%d\n", total_space);
	int devNum;
	hipGetDevice(&devNum);
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, devNum);
	printf("Total global memory:                %lu\n",  devProp.totalGlobalMem);
	printf("Percentage allocated:               %f%%\n", (double)total_space/devProp.totalGlobalMem * 100);
}

